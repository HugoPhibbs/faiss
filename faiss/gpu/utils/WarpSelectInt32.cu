/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/warpselect/WarpSelectImpl.cuh>

/**
 * Custom int32 warp select declarations
 *
 * @author Hugo Phibbs
 */
namespace faiss {
namespace gpu {

WARP_SELECT_DECL(int, true, 1);
WARP_SELECT_DECL(int, false, 1);

WARP_SELECT_DECL(int, true, 32);
WARP_SELECT_DECL(int, false, 32);

WARP_SELECT_DECL(int, true, 64);
WARP_SELECT_DECL(int, false, 64);

WARP_SELECT_DECL(int, true, 128);
WARP_SELECT_DECL(int, false, 128);

WARP_SELECT_DECL(int, true, 256);
WARP_SELECT_DECL(int, false, 256);

WARP_SELECT_DECL(int, true, 512);
WARP_SELECT_DECL(int, false, 512);

WARP_SELECT_DECL(int, true, 1024);
WARP_SELECT_DECL(int, false, 1024);

#if GPU_MAX_SELECTION_K >= 2048
WARP_SELECT_DECL(int, true, 2048);
WARP_SELECT_DECL(int, false, 2048);
#endif

void runWarpSelect(
        Tensor<int, 2, true>& in,
        Tensor<int, 2, true>& outK,
        Tensor<idx_t, 2, true>& outV,
        bool dir,
        int k,
        hipStream_t stream) {
    FAISS_ASSERT(k <= 2048);

    if (dir) {
        if (k == 1) {
            WARP_SELECT_CALL(int, true, 1);
        } else if (k <= 32 && getWarpSizeCurrentDevice() == 32) {
            WARP_SELECT_CALL(int, true, 32);
        } else if (k <= 64) {
            WARP_SELECT_CALL(int, true, 64);
        } else if (k <= 128) {
            WARP_SELECT_CALL(int, true, 128);
        } else if (k <= 256) {
            WARP_SELECT_CALL(int, true, 256);
        } else if (k <= 512) {
            WARP_SELECT_CALL(int, true, 512);
        } else if (k <= 1024) {
            WARP_SELECT_CALL(int, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            WARP_SELECT_CALL(int, true, 2048);
#endif
        }
    } else {
        if (k == 1) {
            WARP_SELECT_CALL(int, false, 1);
        } else if (k <= 32 && getWarpSizeCurrentDevice() == 32) {
            WARP_SELECT_CALL(int, false, 32);
        } else if (k <= 64) {
            WARP_SELECT_CALL(int, false, 64);
        } else if (k <= 128) {
            WARP_SELECT_CALL(int, false, 128);
        } else if (k <= 256) {
            WARP_SELECT_CALL(int, false, 256);
        } else if (k <= 512) {
            WARP_SELECT_CALL(int, false, 512);
        } else if (k <= 1024) {
            WARP_SELECT_CALL(int, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
        } else if (k <= 2048) {
            WARP_SELECT_CALL(int, false, 2048);
#endif
        }
    }
}

} // namespace gpu
} // namespace faiss
