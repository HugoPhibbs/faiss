/*
 * Copyright (c) Meta Platforms, Inc. and affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

 #include <faiss/gpu/utils/DeviceDefs.cuh>
 #include <faiss/gpu/utils/warpselect/WarpSelectImpl.cuh>
 
 #include <cstdint>

 /**
 * Custom long (int64) warp select implementations=
 * 
 * @author Hugo Phibbs
 */
 namespace faiss {
 namespace gpu {
 
 WARP_SELECT_DECL(long, true, 1);
 WARP_SELECT_DECL(long, false, 1);
 
 WARP_SELECT_DECL(long, true, 32);
 WARP_SELECT_DECL(long, false, 32);

 WARP_SELECT_DECL(long, true, 64);
 WARP_SELECT_DECL(long, false, 64);
 
 WARP_SELECT_DECL(long, true, 128);
 WARP_SELECT_DECL(long, false, 128);
 
 WARP_SELECT_DECL(long, true, 256);
 WARP_SELECT_DECL(long, false, 256);
 
 WARP_SELECT_DECL(long, true, 512);
 WARP_SELECT_DECL(long, false, 512);
 
 WARP_SELECT_DECL(long, true, 1024);
 WARP_SELECT_DECL(long, false, 1024);
 
 #if GPU_MAX_SELECTION_K >= 2048
 WARP_SELECT_DECL(long, true, 2048);
 WARP_SELECT_DECL(long, false, 2048);
 #endif

 void runWarpSelect(
    Tensor<long, 2, true>& in,
    Tensor<long, 2, true>& outK,
    Tensor<idx_t, 2, true>& outV,
    bool dir,
    int k,
    hipStream_t stream) {
FAISS_ASSERT(k <= 2048);

if (dir) {
    if (k == 1) {
        WARP_SELECT_CALL(long, true, 1);
    } else if (k <= 32 && getWarpSizeCurrentDevice() == 32) {
        WARP_SELECT_CALL(long, true, 32);
    } else if (k <= 64) {
        WARP_SELECT_CALL(long, true, 64);
    } else if (k <= 128) {
        WARP_SELECT_CALL(long, true, 128);
    } else if (k <= 256) {
        WARP_SELECT_CALL(long, true, 256);
    } else if (k <= 512) {
        WARP_SELECT_CALL(long, true, 512);
    } else if (k <= 1024) {
        WARP_SELECT_CALL(long, true, 1024);
#if GPU_MAX_SELECTION_K >= 2048
    } else if (k <= 2048) {
        WARP_SELECT_CALL(long, true, 2048);
#endif
    }
} else {
    if (k == 1) {
        WARP_SELECT_CALL(long, false, 1);
    } else if (k <= 32 && getWarpSizeCurrentDevice() == 32) {
        WARP_SELECT_CALL(long, false, 32);
    } else if (k <= 64) {
        WARP_SELECT_CALL(long, false, 64);
    } else if (k <= 128) {
        WARP_SELECT_CALL(long, false, 128);
    } else if (k <= 256) {
        WARP_SELECT_CALL(long, false, 256);
    } else if (k <= 512) {
        WARP_SELECT_CALL(long, false, 512);
    } else if (k <= 1024) {
        WARP_SELECT_CALL(long, false, 1024);
#if GPU_MAX_SELECTION_K >= 2048
    } else if (k <= 2048) {
        WARP_SELECT_CALL(long, false, 2048);
#endif
    }
}
}
 
 } // namespace gpu
 } // namespace faiss
 